#include "hip/hip_runtime.h"
#include "common.cuh"
#include "cross-entropy-loss.cuh"
#include "sum.cuh"

#include <cmath>
#include <cstdint>

static __global__ void cross_entropy_loss_f32(const float * logits, const float * labels, float * dst, const int nclasses, const int k) {
    logits += blockIdx.x*nclasses;
    labels += blockIdx.x*nclasses;

    // Find maximum for softmax:
    float max_logit = -INFINITY;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        max_logit = fmaxf(max_logit, logits[i]);
    }
    max_logit = warp_reduce_max(max_logit);

    // Calculate log(softmax(logits)) which is just logits - max:
    float sum = 0.0f;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        sum += expf(logits[i] - max_logit);
    }
    sum = warp_reduce_sum(sum);
    sum = logf(sum);

    // log(exp(logits - max) / sum) = (logits - max) - log(sum)
    float loss = 0.0f;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        loss += (logits[i] - max_logit - sum) * labels[i];
    }
    loss = -warp_reduce_sum(loss) / (float)k;

    if (threadIdx.x != 0) {
        return;
    }

    dst[blockIdx.x] = loss;
}

static __global__ void cross_entropy_loss_back_f32(const float * logits, const float * labels, const float * loss, float * dst, const int nclasses) {
    extern __shared__ float tmp[];

    float maxval = -INFINITY;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        const float val = logits[blockIdx.x*nclasses + i];
        maxval = fmaxf(maxval, val);
        tmp[i] = val;
    }
    maxval = warp_reduce_max(maxval);

    float sum = 0.0f;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        const float val = expf(tmp[i] - maxval);
        sum += val;
        tmp[i] = val;
    }
    sum = warp_reduce_sum(sum);
    const float sm_scale = 1.0f/sum;

    const float d_by_nrows = *loss/gridDim.x;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        dst[blockIdx.x*nclasses + i] = (tmp[i]*sm_scale - labels[blockIdx.x*nclasses + i])*d_by_nrows;
    }
}

void ggml_cuda_cross_entropy_loss(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(dst));

    const int64_t ne00  = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    ggml_cuda_pool & pool = ctx.pool();
    hipStream_t stream = ctx.stream();

    const dim3 blocks_dim(WARP_SIZE, 1, 1);
    const dim3 blocks_num(nrows, 1, 1);
    const int shmem = 0;

    ggml_cuda_pool_alloc<float> dst_tmp(pool, blocks_num.x);

    cross_entropy_loss_f32<<<blocks_num, blocks_dim, shmem, stream>>>(src0_d, src1_d, dst_tmp.ptr, ne00, nrows);
    CUDA_CHECK(hipGetLastError());

    // Combine results from individual blocks:
    sum_f32_cuda(pool, dst_tmp.ptr, dst_d, blocks_num.x, stream);
}

void ggml_cuda_cross_entropy_loss_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const ggml_tensor * opt0 = dst->src[2];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(opt0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(opt0));
    GGML_ASSERT(ggml_is_contiguous(dst));
    GGML_ASSERT(ggml_are_same_shape(src0, src1));
    GGML_ASSERT(ggml_are_same_shape(src0, dst));

    const int64_t ne00  = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    const float * opt0_d = (const float *) opt0->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    const dim3 blocks_dim(WARP_SIZE, 1, 1);
    const dim3 blocks_num(nrows, 1, 1);
    const int shmem = ne00*sizeof(float);

    cross_entropy_loss_back_f32<<<blocks_num, blocks_dim, shmem, stream>>>(src0_d, src1_d, opt0_d, dst_d, ne00);
}
